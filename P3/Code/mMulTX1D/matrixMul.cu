#include "hip/hip_runtime.h"
//
//	matrixMul.cu
//
//	Compile: nvcc matrixMul.cu matrixMul_gold.cpp -O3 -lm -o mMul
//	Usage: ./mMul

// includes, kernels
// #include "mMul.cu"
#include <stdio.h>
#include "matrixMul.h"

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest(int argc, char** argv);
void randomInit(float*, int);
void printDiff(float*, float*, int, int, int, float);
__global__ void
matrixMulKernel( float* Md, float* Nd, float* Pd, int Width);

extern "C"
void computeGold(float*, const float*, const float*, unsigned int);

texture <float> texM;
texture <float> texN;

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{

    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // set seed for rand()
    srand(2006);

    // allocate host memory for matrices M and N
    unsigned int size_M = WIDTH * WIDTH;
    unsigned int mem_size_M = sizeof(float) * size_M;
    float* h_M = (float*)malloc(mem_size_M);
    unsigned int size_N = WIDTH * WIDTH;
    unsigned int mem_size_N = sizeof(float) * size_N;
    float* h_N = (float*)malloc(mem_size_N);

    // initialize host memory
    randomInit(h_M, size_M);
    randomInit(h_N, size_N);

    // allocate device memory
    float* d_M;
    hipMalloc((void**) &d_M, mem_size_M);
    float* d_N;
    hipMalloc((void**) &d_N, mem_size_N);

    hipBindTexture(NULL, texM, d_M, WIDTH*WIDTH*sizeof(float));
    hipBindTexture(NULL, texN, d_N, WIDTH*WIDTH*sizeof(float));

    hipEventRecord(start, 0);

    // copy host memory to device
    hipMemcpy(d_M, h_M, mem_size_M,
                              hipMemcpyHostToDevice);
    hipMemcpy(d_N, h_N, mem_size_N,
                              hipMemcpyHostToDevice);

    // allocate device memory for result
    unsigned int size_P = WIDTH * WIDTH;
    unsigned int mem_size_P = sizeof(float) * size_P;
    float* d_P;
    hipMalloc((void**) &d_P, mem_size_P);

    // allocate host memory for the result
    float* h_P = (float*) malloc(mem_size_P);
    
    // setup execution parameters
    dim3 block(WIDTH/TILE_WIDTH, WIDTH/TILE_WIDTH);
    dim3 threads(TILE_WIDTH, TILE_WIDTH);

    matrixMulKernel<<< block, threads >>>(d_M, d_N, d_P, WIDTH);
    hipDeviceSynchronize();
    
    // copy result from device to host
    hipMemcpy(h_P, d_P, mem_size_P,
                              hipMemcpyDeviceToHost);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("Elapsed time = %f ms\n", time);

    // compute reference solution
    float* reference = (float*)malloc(mem_size_P);
    computeGold(reference, h_M, h_N, WIDTH);

    // check result
    printDiff(reference, h_P, WIDTH, WIDTH, 100, 1.0e-5f);

    // clean up memory
    hipUnbindTexture(texM);
    hipUnbindTexture(texN);
    free(h_M);
    free(h_N);
    free(h_P);
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);

    hipDeviceReset();
}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

void printDiff(float *data1, float *data2, int width, int height, int iListLength, float fListTol)
{
    printf("Listing first %d Differences > %.6f...\n", iListLength, fListTol);
    int j,k;
    int error_count=0;
    for (j = 0; j < height; j++) 
    {
        if (error_count < iListLength)
        {
            printf("\n  Row %d:\n", j);
        }
        for (int i = 0; i < width; i++) 
        {
            k = j * width + i;
            float fDiff = fabs(data1[k] - data2[k]) / data1[k];
            if (fDiff > fListTol) 
            {                
                if (error_count < iListLength)
                {
                    printf("    Loc(%d,%d)\tCPU=%.5f\tGPU=%.5f\tDiff=%.6f\n", i, j, data1[k], data2[k], fDiff);
                }
                error_count++;
            }
        }
    }
    printf(" \n  Total Errors = %d\n\n", error_count);
}

__global__ void
matrixMulKernel( float* Md, float* Nd, float* Pd, int Width)
{

    // Thread index
    float Psub = 0.;
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x*blockDim.x + threadIdx.x;

    for (int k = 0; k < Width; ++k) {
        float m = tex1Dfetch(texM, k + Row * Width);
	float n = tex1Dfetch(texN, Col + k * Width);
        //Psub += Md[Row * Width + k] * Nd[k * Width + Col];
	Psub += m * n;
    }
    Pd[Row * Width + Col] = Psub;

}
