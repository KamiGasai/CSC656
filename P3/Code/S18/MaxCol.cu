#include "hip/hip_runtime.h"
// File: MaxCol.cu
// Compile: nvcc MaxCol.cu -o mc
// Run: ./mc [width of matrix] [threads per block]

// Description: finds the max of each column of a randomly generated matrix
// 		in kernel findMax(), each thread finds the max of one column

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define THREADSPERBLOCK 4

int checkArray(int [], int [], int);

__global__ void findMax(int *m, int *rs, int n);

int main(int argc, char **argv)
{
    /* variables for timing */
    hipEvent_t start, stop;
    float time;

    if (argc != 3) {
       printf("Usage: ./SR [width of matrix] [threads per block]\n");
       exit(0);
    }

    int n = atoi(argv[1]);  // number of matrix rows/cols
    int *hm, // host matrix
        *dm, // device matrix
        *hcs, // host column sums
        *dcs; // device column sums
    int *checkCs;
    int msize = n * n * sizeof(int);  // size of matrix in bytes
    int rssize = n * sizeof(int);
    int threadsPerBlock = atoi(argv[2]); // get threads per block

    if (n % threadsPerBlock != 0) {
       printf("Warning: width of matrix not divisible by # threads per block\n");
    }

    // allocate space for host matrix
    hm = (int *) malloc(msize);  

    // create timer events
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // as a test, fill matrix with random integers

    int i, j;
    for (i = 0; i < n; i++) {
       for (j = 0; j < n; j++) {
          hm[i*n+j] = random() % RAND_MAX;
       }
    }

    // compute max of columns on CPU for checking
    checkCs = (int *) malloc(rssize);
    for (i=0; i<n; i++) {
       checkCs[i] = hm[i];
       for (j=0; j<n; j++) {
          if (checkCs[i] < hm[i + j*n])
             checkCs[i] = hm[i + j*n];
       }
    }

    // allocate space for device matrix 
    hipMalloc((void **)&dm,msize);
    // copy host matrix to device matrix
    hipMemcpy(dm,hm,msize,hipMemcpyHostToDevice);
    // allocate host, device rowsum arrays
    hcs = (int *) malloc(rssize);  
    hipMalloc((void **)&dcs,rssize);

    // record start timestamp
    hipEventRecord(start, 0);

    // invoke the kernel
    findMax<<<n/threadsPerBlock,threadsPerBlock>>>(dm,dcs,n);
    // wait for kernel to finish
    hipDeviceSynchronize();
    // copy row vector from device to host
    hipMemcpy(hcs,dcs,rssize,hipMemcpyDeviceToHost);

    // get elapsed time
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    printf("Elapsed time = %f\n", time);

    // check results
    int diff = checkArray(hcs, checkCs, n);
    if (diff == 0) {
       printf("Arrays match\n");
    }
    else {
       printf("Arrays do not match\n");
    }


    // clean up
    free(hm);
    hipFree(dm);
    free(hcs);
    hipFree(dcs);
}

int checkArray(int x[], int y[], int size) {
   int i;
   int numDiff = 0;
   
   for (i=0; i<size; i++) {
      if (x[i] != y[i]) {
         numDiff++;
      }
   }
   return numDiff;
}

// findMax(int *m, int *cs, int n)
// m: n x n matrix (input)
// cs: cs[i] contains max of columnn i of m (output)
// n: number of elements in each row/column of m

__global__ void findMax(int *m, int *cs, int n)
{
   // your code goes here
}
